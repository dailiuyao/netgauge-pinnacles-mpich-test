#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

struct LogMessage_lyd* d_messages;

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


int main(int argc, char* argv[])
{
  int size = 32*1024*1024;


  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));


  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));


 // Initialize NCCL communication
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  // Create events for timing
  hipEvent_t start, stop;
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  // Temporary buffer to hold timing information
  float *p2pTimes = (float*)malloc((nRanks / 2) * sizeof(float));

  int half_nRanks = nRanks / 2;

  // Communicate from each GPU in node0 to each GPU in node1
  for (int i = 0; i < half_nRanks; ++i) {
      if (myRank == i) { // This process is on node0

          int j = half_nRanks+1;
          // Start timing
          CUDACHECK(hipEventRecord(start, s));

          // Communicating using NCCL
          ncclGroupStart();
          NCCLCHECK(ncclSend((const void*)sendbuff, size, ncclFloat, i, comm, s));
          NCCLCHECK(ncclRecv((void*)recvbuff, size, ncclFloat, j, comm, s));
          ncclGroupEnd();

          // Stop timing
          CUDACHECK(hipEventRecord(stop, s));
          CUDACHECK(hipEventSynchronize(stop));

          // Calculate the elapsed time
          CUDACHECK(hipEventElapsedTime(&p2pTimes[j - half_nRanks], start, stop));
          printf("P2P time between GPU %d on node0 and GPU %d on node1: %f ms\n", i, j - half_nRanks, p2pTimes[j - half_nRanks]);
          
      } 
  }

  // Free the event resources
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));

  // Free the timing buffer
  free(p2pTimes);


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  return 0;
}