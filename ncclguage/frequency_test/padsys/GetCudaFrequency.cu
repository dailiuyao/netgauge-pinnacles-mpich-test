#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipError_t status;
    int device;
    int clockRate;

    // Get the current device
    status = hipGetDevice(&device);
    if (status != hipSuccess) {
        printf("hipGetDevice failed: %s\n", hipGetErrorString(status));
        return 1;
    }

    // Get the clock rate of the current device
    status = hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, device);
    if (status != hipSuccess) {
        printf("hipDeviceGetAttribute failed: %s\n", hipGetErrorString(status));
        return 1;
    }

    // Print the clock rate in kHz (note: to convert to Hz, multiply by 1000)
    printf("GPU Clock Rate: %d kHz\n", clockRate);

    // Optionally, convert to Hz for calculations
    float clockRateHz = clockRate * 1000.0f;
    printf("GPU Clock Rate: %.0f Hz\n", clockRateHz);

    return 0;
}
