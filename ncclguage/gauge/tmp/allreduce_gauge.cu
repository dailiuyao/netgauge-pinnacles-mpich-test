#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

struct LogMessage_lyd_global* d_messages;
// int nccl_gauge_iteration = 0;

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

uint64_t rdtsc() {
    uint32_t lo, hi;
    // Inline assembly to read the TSC
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return (uint64_t)hi << 32 | lo;
}


int main(int argc, char* argv[])
{

  const char* env_gauge_heo_var = getenv("GAUGE_HEO");

  const char* env_gauge_mode_var = getenv("GAUGE_MODE");

  const char* env_gauge_iteration_var = getenv("GAUGE_ITERATION");

  const char* env_gauge_nchannels_var = getenv("GAUGE_NCHANNELS");

  const char* env_gauge_chunk_size_var = getenv("GAUGE_CHUNK_SIZE");

  const char* env_gauge_output_dir_var = getenv("GAUGE_OUT_DIRE");

  // Check if environment variables are set
  if (!env_gauge_heo_var) env_gauge_heo_var = "unknown_gauge_heo";
  if (!env_gauge_mode_var) env_gauge_mode_var = "unknown_gauge_mode";
  if (!env_gauge_iteration_var) env_gauge_iteration_var = "unknown_gauge_iteration";
  if (!env_gauge_nchannels_var) env_gauge_nchannels_var = "unknown_gauge_nchannels";
  if (!env_gauge_chunk_size_var) env_gauge_chunk_size_var = "unknown_gauge_chunk_size";
  if (!env_gauge_output_dir_var) {
    env_gauge_output_dir_var = "unknown_gauge_output_dir";
    printf("unknown gauge output dir\n");
  }


  int size = 1;  // Default size
  const char* env_gauge_size_var = getenv("GAUGE_MESSAGE_SIZE");
  if (env_gauge_size_var != nullptr) {
      size = atoi(env_gauge_size_var) * 1024 / 4;  // Convert from kilobytes to number of floats, assuming the environment variable is in kilobytes
  }


  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  char filename[256];

  if (myRank < 2) {
    sprintf(filename, "%s/nccl_pping_%s_chunk%s_itr0-5-r%d.out", env_gauge_output_dir_var, env_gauge_heo_var, env_gauge_chunk_size_var, myRank);
    freopen(filename, "a", stdout);
  } else {
    freopen("/dev/null", "w", stdout);
  }


  // int nccl_start = 0;
  // int nccl_end = 0;

  // nccl_start = clock();

  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));
  

  //gauge test
  CUDACHECK(hipMalloc(&d_messages, sizeof(LogMessage_lyd_global)));
  CUDACHECK(hipMemset(d_messages, 0, sizeof(LogMessage_lyd_global))); 

  // // Declare CUDA events
  // hipEvent_t start_0, stop_0;
  // hipEventCreate(&start_0);
  // hipEventCreate(&stop_0);
  // float milliseconds_0 = 0;

  // hipEventRecord(start_0, s);

  uint64_t kernel_gauge_start = rdtsc();


  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  //communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum,
        comm, s));

  // hipEventRecord(stop_0, s);

  // hipEventSynchronize(stop_0);

  // hipEventElapsedTime(&milliseconds_0, start_0, stop_0);

  // printf("heo(%s)_mode(%s)_nchannels(%s)_chunk size(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_chunk_size_var, env_gauge_size_var, N_ITERS, env_gauge_iteration_var, milliseconds_0/1.44e3);


  // // Clean up
  // hipEventDestroy(start_0);
  // hipEventDestroy(stop_0);


  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  uint64_t kernel_gauge_end = rdtsc();

  // After the kernel execution, copy the messages back to the host
  LogMessage_lyd_global* h_messages = new LogMessage_lyd_global;
  hipMemcpy(h_messages, d_messages, sizeof(LogMessage_lyd_global), hipMemcpyDeviceToHost);

  // Process and print the messages on the host
  #if PROFILE_LYD_REDUCE_BROADCAST == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | allreduce.h | runTreeUpDown | recvReduceCopy | time: %f us\n", h_messages->timeValue[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | allreduce.h | runTreeUpDown | directSendFromOutput | time: %f us\n", h_messages->timeValue1[i][0]);
  }
  #endif

  #if PROFILE_LYD_REDUCE_BROADCAST_CHUNK == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | allreduce.h | runTreeUpDown | recvReduceCopy-chunk | iteration %d | time: %f us\n", j, h_messages->timeValue[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | allreduce.h | runTreeUpDown | directSendFromOutput-chunk | iteration %d | time: %f us\n", j, h_messages->timeValue1[i][j]);
    }
  }
  #endif

  #if PROFILE_LYD_REDUCE_LOADCONN_SETDATA == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | loadRecvConn | time: %f us\n", h_messages->timeValue[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | loadSendConn | time: %f us\n", h_messages->timeValue1[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | setDataPtrs | time: %f us\n", h_messages->timeValue2[i][0]);
  }
  #endif

  #if PROFILE_LYD_GENERIC == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | genericop | time: %f us\n", h_messages->timeValue[i][0]);
  }
  #endif

  #if PROFILE_LYD_WAIT_REDUCE_COPY_POST == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | waitpeer | iteration %d | time: %f us\n", j, h_messages->timeValue[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | ReduceOrCopyMulti | iteration %d | time: %f us\n", j, h_messages->timeValue1[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | postPeer | iteration %d | time: %f us\n", j, h_messages->timeValue2[i][j]);
    }
  }
  #endif

  #if PROFILE_LYD_SEND_RECV_CHUNK == 1
  double gauge_time;

  if (myRank == 0) { 
    gauge_time = static_cast<double>(h_messages->timeValue[1][0] - h_messages->timeValue[0][0]) / 1410.0;
    printf("heo(%s)_mode(%s)_nchannels(%s)_chunk size(%s)_message size(%s)_n(%d)_d(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_chunk_size_var, env_gauge_size_var, N_ITERS, GAUGE_D, env_gauge_iteration_var, gauge_time);
    printf("nccl kernel elapsed time: %f us\n", static_cast<double>(kernel_gauge_end - kernel_gauge_start) / 2800.0);
  } else {
    gauge_time = static_cast<double>(h_messages->timeValue[0][0] - h_messages->timeValue[1][0]) / 1410.0; 
    printf("heo(%s)_mode(%s)_nchannels(%s)_chunk size(%s)_message size(%s)_n(%d)_d(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_chunk_size_var, env_gauge_size_var, N_ITERS, GAUGE_D, env_gauge_iteration_var, gauge_time);
  }

  // print the gap between chunks
  if (myRank == 0) { 
    for (size_t i = 1; i < N_ITERS; ++i) {
      gauge_time = static_cast<double>(h_messages->timeValue[0][i] - h_messages->timeValue[0][0]) / 1410.0;
      printf("chunk gap | chunk%d -> chunk0 | heo(%s)_mode(%s)_nchannels(%s)_chunk size(%s)_message size(%s)_n(%d)_d(%d)_iteration(%s): %f us\n", i, env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_chunk_size_var, env_gauge_size_var, N_ITERS, GAUGE_D, env_gauge_iteration_var, gauge_time);
    }
  } else {
    for (size_t i = 1; i < N_ITERS; ++i) {
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[1][0]) / 1410.0; 
      printf("chunk gap | chunk%d -> chunk0 | heo(%s)_mode(%s)_nchannels(%s)_chunk size(%s)_message size(%s)_n(%d)_d(%d)_iteration(%s): %f us\n", i, env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_chunk_size_var, env_gauge_size_var, N_ITERS, GAUGE_D, env_gauge_iteration_var, gauge_time);
    }
  }
  #endif

  // Free the device memory of the gauge test
  hipFree(d_messages);
  delete[] h_messages;


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);

  // nccl_end = clock();

  // printf("heo(%s)_mode(%s)_nchannels(%s)_chunk size(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_chunk_size_var, env_gauge_size_var, N_ITERS, env_gauge_iteration_var, (nccl_end - nccl_start)/1.44e3);

  //finalizing MPI
  MPICHECK(MPI_Finalize());


  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}