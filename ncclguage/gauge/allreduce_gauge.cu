#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

struct LogMessage_lyd* d_messages;

std::chrono::time_point<std::chrono::high_resolution_clock> netIsend_time_start[MAXLOGLYD];
std::chrono::time_point<std::chrono::high_resolution_clock> netIrecv_time_start[MAXLOGLYD];

// #define WARMUP_ITERATION 5
#define WARMUP_SIZE 32
#define DEFAULT_D 0

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

uint64_t rdtsc() {
    uint32_t lo, hi;
    // Inline assembly to read the TSC
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return (uint64_t)hi << 32 | lo;
}

void busyWaitMilliseconds(int ms) {
    auto start = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::milliseconds(ms);

    // Spin in a loop until the desired time has elapsed
    while (std::chrono::high_resolution_clock::now() - start < duration) {
        // Do nothing
    }
}



int main(int argc, char* argv[])
{

  const char* env_gauge_heo_var = getenv("GAUGE_HEO");

  const char* env_gauge_mode_var = getenv("GAUGE_MODE");

  const char* env_gauge_iteration_var = getenv("GAUGE_ITERATION");

  const char* env_gauge_nchannels_var = getenv("GAUGE_NCHANNELS");

  const char* env_gauge_chunk_size_var = getenv("GAUGE_CHUNK_SIZE");

  const char* env_gauge_output_dir_var = getenv("GAUGE_OUT_DIRE");

  const char* env_gauge_nthreads_var = getenv("NCCL_NTHREADS");

  const char* env_comm_gpu_id_var = getenv("COMM_GPU_ID");

  const char* env_experiment_id_var = getenv("GAUGE_EXPERIMENT_ID");

  const char* env_gauge_algo_var = getenv("NCCL_ALGO");


  // Check if environment variables are set
  if (!env_gauge_heo_var) env_gauge_heo_var = "unknown_gauge_heo";
  if (!env_gauge_mode_var) env_gauge_mode_var = "unknown_gauge_mode";
  if (!env_gauge_iteration_var) env_gauge_iteration_var = "unknown_gauge_iteration";
  if (!env_gauge_nchannels_var) env_gauge_nchannels_var = "unknown_gauge_nchannels";
  if (!env_gauge_chunk_size_var) env_gauge_chunk_size_var = "unknown_gauge_chunk_size";
  if (!env_gauge_nthreads_var) env_gauge_nthreads_var = "unknown_gauge_nthreads";  
  if (!env_gauge_output_dir_var) {
    env_gauge_output_dir_var = "unknown_gauge_output_dir";
    printf("unknown gauge output dir\n");
  }

  long long size = 1;  // Default size
  const char* env_gauge_size_var = getenv("GAUGE_MESSAGE_SIZE");
  if (env_gauge_size_var != nullptr) {
      size = atoll(env_gauge_size_var) * 1024 / 4;  // Convert from kilobytes to number of floats, assuming the environment variable is in kilobytes
  }

  const char* env_gauge_step_var = getenv("GAUGE_STEP_SIZE");

  int gauge_step = atoi(env_gauge_step_var);

  int comm_gpu_id = atoi(env_comm_gpu_id_var);

  int gauge_d = DEFAULT_D;

  gauge_d = atoi(argv[1]);

  int N_CHUNKS;

  // if (gauge_step != 0) {
  //   if (gauge_step >= 16384) {
  //     N_CHUNKS = 128;
  //   } else {
  //     N_CHUNKS = atoi(env_gauge_size_var)/atoi(env_gauge_step_var); 
  //   }
  // } else {
  //   N_CHUNKS = 1;
  // }

  // if (N_CHUNKS == 0) N_CHUNKS = 1;

  N_CHUNKS = MAXLOGLYD - 1;

  int myRank, nRanks, localRank = 0;

  // Set the device scheduling flag before creating a device context
    hipError_t err = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set device flags: %s\n", hipGetErrorString(err));
        return 1;
    }

  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  char filename[256];

  if (myRank == 0 || myRank == nRanks / 2 || myRank == nRanks / 2 - 1 || myRank == nRanks / 2 + 1) {
    sprintf(filename, "%s/nccl_allreduce_%s_chunk-%s_r-%d_e-%s.out", env_gauge_output_dir_var, env_gauge_heo_var, env_gauge_chunk_size_var, myRank, env_experiment_id_var);
    freopen(filename, "a", stdout);
  } else {
    freopen("/dev/null", "w", stdout);
  }

  // printf("proc is: %d\n", myRank);
  // int gdb = 1;
  // if (myRank == 0){
  //   gdb = 0;
  //   printf("proc is: %d, pid is %d\n", myRank, (int)getpid());
  // }
  // while(gdb == 0){
  //   printf("loop\n");
  //   sleep(10);
  // }

  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }

  // select gpu on each node
  if (comm_gpu_id == 0) {
    localRank = localRank + 1;
  } else if (comm_gpu_id == 1){
    if (myRank == 0) localRank = localRank + 1;
  } else if (comm_gpu_id == 2){
    if (myRank == 1) localRank = localRank + 1;
  } else if (comm_gpu_id == 3){
    localRank = localRank + 3;
  } 


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, WARMUP_ITERATION * WARMUP_SIZE * sizeof(float) + N_ITERS * size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, WARMUP_ITERATION * WARMUP_SIZE * sizeof(float) + N_ITERS * size * sizeof(float))); 
  CUDACHECK(hipStreamCreate(&s));
  

  //gauge test
  LogMessage_lyd host_messages;
  // Initialize other members of host_messages as needed
  memset(&host_messages, 0, sizeof(LogMessage_lyd)); // Zero-initialize the struct
  host_messages.gauge_d = gauge_d;
  host_messages.gauge_iteration = WARMUP_ITERATION+N_ITERS;

  CUDACHECK(hipMalloc(&d_messages, sizeof(LogMessage_lyd)));

  CUDACHECK(hipMemcpy(d_messages, &host_messages, sizeof(LogMessage_lyd), hipMemcpyHostToDevice));

  
  ////////////////////////////// PROFILE_LYD_AllReduce_DEVICE: START //////////////////////////////
  
  #if PROFILE_LYD_REDUCE_BROADCAST_CHUNK == 1
  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  //communicating using NCCL

  hipEvent_t start, stop;
  float elapsed_time;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i=0; i < N_ITERS; i++){
    netIsend_time_start[i] = std::chrono::high_resolution_clock::now(); 
    netIrecv_time_start[i] = std::chrono::high_resolution_clock::now(); 
  }

  // Warm up START 
  CUDACHECK(hipStreamSynchronize(s));

  for (int i = 0 ; i < WARMUP_ITERATION; i++) {
    NCCLCHECK(ncclAllReduce((const void*)((float*)sendbuff + i * WARMUP_SIZE), 
                        (void*)((float*)recvbuff + i * WARMUP_SIZE), 
                        WARMUP_SIZE, 
                        ncclFloat, 
                        ncclSum, 
                        comm, 
                        s));
  }

  CUDACHECK(hipStreamSynchronize(s));

  // Warm up END

  hipEventRecord(start, s);

  std::chrono::time_point<std::chrono::high_resolution_clock> nccl_func_start_time = std::chrono::high_resolution_clock::now(); 

  CUDACHECK(hipStreamSynchronize(s));

  for (int i = 0; i < N_ITERS; i++) {
    NCCLCHECK(ncclAllReduce((const void*)((float*)sendbuff + i * size), 
                            (void*)((float*)recvbuff + i * size), 
                            size, 
                            ncclFloat, 
                            ncclSum, 
                            comm, 
                            s));
    // busyWaitMilliseconds(gauge_d); 
    CUDACHECK(hipStreamSynchronize(s));
  }
  CUDACHECK(hipStreamSynchronize(s));

  hipEventRecord(stop, s);

  std::chrono::time_point<std::chrono::high_resolution_clock> nccl_func_end_time = std::chrono::high_resolution_clock::now();

  // Wait for the stop event to complete
  hipEventSynchronize(stop);

  // Calculate elapsed time between events
  hipEventElapsedTime(&elapsed_time, start, stop);

  // Destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop); 

  std::chrono::duration<float, std::milli> nccl_func_time = nccl_func_end_time - nccl_func_start_time; 

  #endif

  ////////////////////////////// PROFILE_LYD_AllReduce_DEVICE: END //////////////////////////////

  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  // After the kernel execution, copy the messages back to the host
  LogMessage_lyd* h_messages = new LogMessage_lyd;
  hipMemcpy(h_messages, d_messages, sizeof(LogMessage_lyd), hipMemcpyDeviceToHost);

  #if PROFILE_LYD_REDUCE_BROADCAST_CHUNK == 1
  double gauge_time;
  
  // if (strcmp(env_gauge_algo_var, "Tree") == 0)
  if (myRank == 0) {
    printf("INFO: heo(%s)_mode(%s)_message size(%s)_nchannels(%s)_nthreads(%s)_nmessages(%d)_d(%d)_iteration(%s)\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, gauge_d, env_gauge_iteration_var);
    printf("--nccl allreduce elapsed time by cuda event: %f ms\n", elapsed_time);
    printf("--nccl allreduce elapsed time by clock: %.3f ms\n", nccl_func_time.count());
    for (size_t i = 0; i < N_ITERS; ++i) {
      gauge_time = static_cast<double>(h_messages->timeEndValue[0][i+WARMUP_ITERATION] - h_messages->timeStartValue[0][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d recvReduceCopy time: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeEndValue[1][i+WARMUP_ITERATION] - h_messages->timeStartValue[1][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d directSendFromOutput time: %f us\n", i, gauge_time);
    }
  } else if (myRank == nRanks / 2) {
    printf("INFO: heo(%s)_mode(%s)_message size(%s)_nchannels(%s)_nthreads(%s)_nmessages(%d)_d(%d)_iteration(%s)\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, gauge_d, env_gauge_iteration_var);
    printf("--nccl allreduce elapsed time by cuda event: %f ms\n", elapsed_time);
    printf("--nccl allreduce elapsed time by clock: %.3f ms\n", nccl_func_time.count());
    for (size_t i = 0; i < N_ITERS; ++i) {
      gauge_time = static_cast<double>(h_messages->timeEndValue[0][i+WARMUP_ITERATION] - h_messages->timeStartValue[0][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d recvReduceSend time: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeEndValue[1][i+WARMUP_ITERATION] - h_messages->timeStartValue[1][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d directRecvCopySend time: %f us\n", i, gauge_time);
    }
  } else if (myRank == nRanks / 2 - 1) {
    printf("INFO: heo(%s)_mode(%s)_message size(%s)_nchannels(%s)_nthreads(%s)_nmessages(%d)_d(%d)_iteration(%s)\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, gauge_d, env_gauge_iteration_var);
    printf("--nccl allreduce elapsed time by cuda event: %f ms\n", elapsed_time);
    printf("--nccl allreduce elapsed time by clock: %.3f ms\n", nccl_func_time.count());
    for (size_t i = 0; i < N_ITERS; ++i) {
      gauge_time = static_cast<double>(h_messages->timeEndValue[0][i+WARMUP_ITERATION] - h_messages->timeStartValue[0][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d send time: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeEndValue[1][i+WARMUP_ITERATION] - h_messages->timeStartValue[1][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d directRecv time: %f us\n", i, gauge_time);
    }
  } else if (myRank == nRanks / 2 + 1) {
    printf("INFO: heo(%s)_mode(%s)_message size(%s)_nchannels(%s)_nthreads(%s)_nmessages(%d)_d(%d)_iteration(%s)\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, gauge_d, env_gauge_iteration_var);
    printf("--nccl allreduce elapsed time by cuda event: %f ms\n", elapsed_time);
    printf("--nccl allreduce elapsed time by clock: %.3f ms\n", nccl_func_time.count());
    for (size_t i = 0; i < N_ITERS; ++i) {
      gauge_time = static_cast<double>(h_messages->timeEndValue[0][i+WARMUP_ITERATION] - h_messages->timeStartValue[0][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d send time: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeEndValue[1][i+WARMUP_ITERATION] - h_messages->timeStartValue[1][i+WARMUP_ITERATION]) / GAUGE_GPU_FREQUENCY;
      printf("--message %d directRecv time: %f us\n", i, gauge_time);
    }
  }

  // print the gap between chunks
  if (myRank == 0) { 
    for (size_t i = 1; i < min(static_cast<size_t>(h_messages->signal[0]), static_cast<size_t>(N_CHUNKS)); ++i) {
      gauge_time = static_cast<double>(h_messages->timeValue[0][i] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d recvReduceCopy - chunk0 recvReduceCopy start: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d directSendFromOutput - chunk0 directSendFromOutput: %f us\n", i, gauge_time);
    }
    gauge_time = static_cast<double>(h_messages->timeValue[0][N_CHUNKS] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d recvReduceCopy - chunk0 recvReduceCopy: %f us\n", h_messages->signal[0], gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[1][N_CHUNKS] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d directSendFromOutput - chunk0 directSendFromOutput: %f us\n", h_messages->signal[0], gauge_time);
  } else if (myRank == nRanks / 2) {
    for (size_t i = 1; i < min(static_cast<size_t>(h_messages->signal[0]), static_cast<size_t>(N_CHUNKS)); ++i) {
      gauge_time = static_cast<double>(h_messages->timeValue[0][i] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d recvReduceSend - chunk0 recvReduceSend: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d directRecvCopySend - chunk0 directRecvCopySend: %f us\n", i, gauge_time);
    }
    gauge_time = static_cast<double>(h_messages->timeValue[0][N_CHUNKS] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d recvReduceSend - chunk0 recvReduceSend: %f us\n", h_messages->signal[0], gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[1][N_CHUNKS] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d directRecvCopySend - chunk0 directRecvCopySend: %f us\n", h_messages->signal[0], gauge_time);
  } else if (myRank == nRanks / 2 - 1) {
    for (size_t i = 1; i < min(static_cast<size_t>(h_messages->signal[0]), static_cast<size_t>(N_CHUNKS)); ++i) {
      gauge_time = static_cast<double>(h_messages->timeValue[0][i] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d send - chunk0 send: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d directRecv - chunk0 directRecv: %f us\n", i, gauge_time);
    }
    gauge_time = static_cast<double>(h_messages->timeValue[0][N_CHUNKS] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d send - chunk0 send: %f us\n", h_messages->signal[0], gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[1][N_CHUNKS] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d directRecv - chunk0 directRecv: %f us\n", h_messages->signal[0], gauge_time);
  } else if (myRank == nRanks / 2 + 1) {
    for (size_t i = 1; i < min(static_cast<size_t>(h_messages->signal[0]), static_cast<size_t>(N_CHUNKS)); ++i) {
      gauge_time = static_cast<double>(h_messages->timeValue[0][i] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d send - chunk0 send: %f us\n", i, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
      printf("--chunk gap | chunk%d directRecv - chunk0 directRecv: %f us\n", i, gauge_time);
    }
    gauge_time = static_cast<double>(h_messages->timeValue[0][N_CHUNKS] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d send - chunk0 send: %f us\n", h_messages->signal[0], gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[1][N_CHUNKS] - h_messages->timeValue[1][0]) / GAUGE_GPU_FREQUENCY;
    printf("--chunk gap | chunk%d directRecv - chunk0 directRecv: %f us\n", h_messages->signal[0], gauge_time);
  }
  #endif

  // Free the device memory of the gauge test
  hipFree(d_messages);
  delete[] h_messages;


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);

  //finalizing MPI
  MPICHECK(MPI_Finalize());

  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}